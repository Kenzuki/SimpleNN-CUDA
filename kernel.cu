#include "hip/hip_runtime.h"


#include <sstream> 
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <time.h>
#include "api.h"
using namespace std;



int main()
{
	srand(time(NULL));
	a();


#pragma region kernel
/*





	host_vector<float> jedn;

	int counter = 0;

	for (int i = 0; i < 200; i++)
	{
		for (int k = 0; k < 10; k++)
		{
			if(k == counter)jedn.push_back(1);
			else jedn.push_back(0);
		}
		counter++;
		if (counter == 10) counter = 0;
	}


	string N1 = "liczby/rzad";
	string N2 = "liczba";
	string N3 = ".bmp";


	Matrix liczby[5000];
	host_vector<float> matrixValues;
	host_vector<float> matrixValues2(400);
	host_vector<float> targets;
	int value = 9;

	// 0- 0000 0000 01
	//1 - 0000 0000 10
	int ile_wczyt = 5000;
	int tab[10] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 1 };
	for (int i = 0; i < ile_wczyt; i++)
	{

		if (i % 500 == 0 && i != 0)
		{
			tab[value] = 0;
			value--;
			tab[value] = 1;
		}

		for (int kk = 0; kk < 10; kk++)
			targets.push_back(tab[kk]);

		char adres[260];

		stringstream liczba;
		liczba << i;
		string liczba_zamieniona = liczba.str();

		stringstream liczba2;
		liczba2 << int(i / 100);
		string liczba_zamieniona2 = liczba2.str();


		string wynik = N1 + liczba_zamieniona2 + N2 + liczba_zamieniona + N3;
		strcpy(adres, wynik.c_str());
		//cout << adres<<endl;
		matrixValues2 = rozpocznij_konw(adres);
		for (int k = 0; k < matrixValues2.size(); k++)
		{
			matrixValues.push_back(matrixValues2[k]);
		}
	}

	aranzacja(targets, matrixValues);

	if (0)
	{
		ofstream plik("wynik.txt", std::ios::out | ios::app);
		ofstream plik2("Macierz.txt", std::ios::out | ios::app);
		ofstream plik3("test.txt", std::ios::out | ios::app);
		//plik.open("wykres.txt", );
		host_vector<float> matrixValuesX(400);
		matrixValuesX = rozpocznij_konw("C:/Users/Patryk/source/repos/neural_network/neural_network/liczby/rzad6liczba657.bmp");

		string x="";	//Wklejamy go do nowego pliku
		for (int z = 0; z<targets.size(); z++)
		{

			stringstream ss;
			string h;
			ss << targets[z];
			h = ss.str();
			x += h;
			if(z%10!=0 || z==0)
			x += ",";
			if (z % 10 == 0 && z != 0)
			{
				x += "\n";

				plik.write(x.c_str(), x.length());
				x = "";
			}


		}
		x = "";
		for (int z = 0; z<matrixValues.size(); z++)
		{

			stringstream ss;
			string h;
			ss << matrixValues[z];
			h = ss.str();
			x += h;
			if (z % 400 != 0 || z == 0)
				x += ",";

			if (z % 400 == 0 && z != 0)
			{
				x += "\n";

				plik2.write(x.c_str(), x.length());
				x = "";
			}


		}
		for (int z = 0; z<matrixValuesX.size(); z++)
		{

			stringstream ss;
			string h;
			ss << matrixValuesX[z];
			h = ss.str();
			x += h;
			if (z % 400 != 0 && z != 0)
				x += ",";

			if (z % 400 == 0 && z != 0)
			{
				x += "\n";

				plik3.write(x.c_str(), x.length());
				x = "";
			}


		}


		


		plik.close();
		plik2.close();
		plik3.close();
	}
		
	//std::system("pause");


	Matrix X = Matrix(ile_wczyt, 400, matrixValues);
	Matrix Y = Matrix(ile_wczyt, 10, targets);
	//Y.PrintMatrix();
	

	vector<int> size = { 35, 10 };

	vector<float> layerA = { float(-1/sqrt(ile_wczyt)),  float(1 / sqrt(ile_wczyt)) };
	//vector<float> layerB = { 0, 1 };
	vector<float> output = { float(-1 / sqrt(ile_wczyt)),  float(1 / sqrt(ile_wczyt)) };
	vector<vector<float>> range = { layerA,output };
	vector<string> activation = { "tanh","sigmoid" };

	//vector<float> temp = { 1,0,1,0,1,0,1,1,0,1,0,1};d
	//host_vector<float> x = temp;
	//vector<float> temp2 = { 1,0,1,0,0,1};
	//host_vector<float> y = temp2;

	//Matrix X = Matrix(3, 4, x);
	//Matrix Y = Matrix(3, 2, y);

	NeuralNetwork net = NeuralNetwork(size, range, activation);
	net.FitData(X);
	auto start = std::chrono::system_clock::now();

	net.Train(Y);

	auto end = std::chrono::system_clock::now();
	std::chrono::duration<double> czas = end - start;

	printf("Uplynelo czasu na cpu :%f \n", czas);
	std::system("pause");

	host_vector<float> matrixValuesX(400);
	matrixValuesX = rozpocznij_konw("C:/Users/Patryk/source/repos/neural_network/neural_network/liczby/rzad6liczba657.bmp");
	cout << endl;
	for (int k = 0; k <400; k++)
	{

		if (k % 20 == 0 && k != 0)
			cout << endl;

		if (matrixValuesX[k] == 1)
			cout << "* ";

		if (matrixValuesX[k] == 0)
			cout << ". ";
		//if (k == 19)
		//	cout << endl;

		
	}

	cout << endl;
	Matrix test = Matrix(1, 400, matrixValuesX);
	
	Matrix result = net.Predict(test);
	cout << endl;
	result.PrintMatrix();
	host_vector<float> u = result.GetAllMatrixValues();

    int wynik = 0;
	float wartosc = u[0];
	for (int y = 1; y < 10; y++)
	{
	
		if (u[y] > wartosc)
		{
			
			wartosc = u[y];
			wynik = y;
		}
	}
	cout << "WYNIK :" << wynik << endl;

	matrixValuesX = rozpocznij_konw("tescik.bmp");
	cout << endl;
	for (int k = 0; k <400; k++)
	{

		if (k % 20 == 0 && k != 0)
			cout << endl;

		if (matrixValuesX[k] == 1)
			cout << "* ";

		if (matrixValuesX[k] == 0)
			cout << ". ";
		//if (k == 19)
		//	cout << endl;


	}

	cout << endl;
	 test = Matrix(1, 400, matrixValuesX);

	 result = net.Predict(test);
	cout << endl;
	result.PrintMatrix();
	 u = result.GetAllMatrixValues();

	 wynik = 0;
	 wartosc = u[0];
	for (int y = 1; y < 10; y++)
	{

		if (u[y] > wartosc)
		{

			wartosc = u[y];
			wynik = y;
		}
	}
	cout << "WYNIK :" << wynik << endl;

	host_vector<float> matrixValues11;
	host_vector<float> matrixValues22(400);
	host_vector<float> targets11;

	int value2 = 9;

	// 0- 0000 0000 01
	//1 - 0000 0000 10
	
	int tab2[10] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 1 };

	for (int i = 0; i < ile_wczyt; i++)
	{

		if (i % 500 == 0 && i != 0)
		{
			tab2[value] = 0;
			value2--;
			tab2[value] = 1;
		}

		for (int kk = 0; kk < 10; kk++)
			targets11.push_back(tab[kk]);

		char adres[260];

		stringstream liczba;
		liczba << i;
		string liczba_zamieniona = liczba.str();

		stringstream liczba2;
		liczba2 << int(i / 100);
		string liczba_zamieniona2 = liczba2.str();


		string wynik = N1 + liczba_zamieniona2 + N2 + liczba_zamieniona + N3;
		strcpy(adres, wynik.c_str());
		//cout << adres<<endl;
		matrixValues22 = rozpocznij_konw(adres);
		for (int k = 0; k < matrixValues22.size(); k++)
		{
			matrixValues11.push_back(matrixValues22[k]);
		}
	}



	Matrix test2 = Matrix(1, 400, matrixValues11);

	//aranzacja(targets, matrixValues);
	X = Matrix(ile_wczyt, 400, matrixValues);
	Y = Matrix(ile_wczyt, 10, targets);

	Matrix result2 = net.Predict(X);
	cout << endl;
	u = result2.GetAllMatrixValues();
	host_vector<float> res;

	for (int i = 0; i < result2.GetRows();i++)
	{
		wynik = 0;
		wartosc = u[i*result2.GetColumns()];
		for (int y = 1; y < 10; y++)
		{
			if (u[i*result2.GetColumns()+y] > wartosc)
			{

				wartosc = u[i*result2.GetColumns() + y];
				wynik = y;
			}
		}
		res.push_back(wynik);
		cout << "WYNIK przewidywany :" << wynik << endl;
	}

	host_vector<float> ys = Y.GetAllMatrixValues();
	host_vector<float> tar;

	for (int i = 0; i < Y.GetRows(); i++)
	{
		wynik = 0;
		wartosc = ys[i*result2.GetColumns()];
		for (int y = 1; y < 10; y++)
		{
			if (ys[i*result2.GetColumns() + y] > wartosc)
			{

				wartosc = ys[i*result2.GetColumns() + y];
				wynik = y;
			}
		}
		tar.push_back(wynik);
		cout << "WYNIK prawdziwy:" << wynik << endl;
	}


	int good = 0;
	int all = result2.GetRows();

	for (int i = 0; i < res.size(); i++)
	{
		if (res[i] == tar[i]) good++;
	}

	float acc = float(100) * float(good) / float(all);
	cout << "Wynik w %: " << acc << endl;






*/
#pragma endregion
	std::system("pause");

}

