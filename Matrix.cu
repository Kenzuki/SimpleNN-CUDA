#include "hip/hip_runtime.h"
#include "Matrix.h"
#include "iostream"

#define NUMBER_OF_THREADS 1024

Matrix::Matrix(int size, host_vector<float> values) {
	this->numberOfRows = size;
	this->numberOfColumns = size;
	this->numberOfElements = size * size;
	AttributeValuesToMatrix(values);
}
Matrix::Matrix(int rows, int columns) {
	this->numberOfRows = rows;
	this->numberOfColumns = columns;
	this->numberOfElements = rows * columns;
	AttributeOnesToMatrix();
}

Matrix::Matrix(int rows, int columns, host_vector<float> &values) {
	this->numberOfRows = rows;
	this->numberOfColumns = columns;
	this->numberOfElements = rows * columns;
	ChangeAllValuesInMatrix(values);
}

Matrix::~Matrix() {}

void Matrix::operator=(Matrix matrix) {
	this->SetRows(matrix.GetRows());
	this->SetColumns(matrix.GetColumns());
	this->ChangeAllValuesInMatrix(matrix.GetAllMatrixValues());
}
Matrix Matrix::operator*(float value) {
	Matrix outMatrix(this->GetRows(), this->GetColumns());

	host_vector<float> h_inMatrixValues = this->GetAllMatrixValues();
	host_vector<float> h_outMatrixValues;
	device_vector<float> d_inMatrixValues = h_inMatrixValues;
	device_vector<float> d_outMatrixValues = h_inMatrixValues;

	float *d_in_pointer = raw_pointer_cast(d_inMatrixValues.data());
	float *d_out_pointer = raw_pointer_cast(d_outMatrixValues.data());

	int blocks = this->GetRows() / NUMBER_OF_THREADS;
	blocks += 1;

	cudaMultiplicationScalar<<<blocks, NUMBER_OF_THREADS>>>(d_in_pointer, d_out_pointer, value, this->GetRows(), this->GetColumns());

	hipDeviceSynchronize();

	h_outMatrixValues = d_outMatrixValues;
	outMatrix.ChangeAllValuesInMatrix(h_outMatrixValues);

	return outMatrix;
}
Matrix Matrix::operator*(Matrix matrix) {
	Matrix outMatrix(this->GetRows(), matrix.GetColumns());

	host_vector<float> h_inMatrixA = this->GetAllMatrixValues();
	host_vector<float> h_inMatrixB = matrix.GetAllMatrixValues();
	host_vector<float> h_outMatrix = outMatrix.GetAllMatrixValues();
	device_vector<float> d_inMatrixA = h_inMatrixA;
	device_vector<float> d_inMatrixB = h_inMatrixB;
	device_vector<float> d_outMatrix = h_outMatrix;

	float *d_in_pointerA = raw_pointer_cast(d_inMatrixA.data());
	float *d_in_pointerB = raw_pointer_cast(d_inMatrixB.data());
	float *d_out_pointer = raw_pointer_cast(d_outMatrix.data());

	//int blocks = this->GetRows() / NUMBER_OF_THREADS;
	//blocks += 1;
	int numberOfElements = this->GetRows()*matrix.GetColumns();

	int blocks = numberOfElements / NUMBER_OF_THREADS;
	blocks++;
	int threads = numberOfElements / blocks;

	cudaMultiplicationMatrix2 << <blocks, threads >> >(d_in_pointerA, d_in_pointerB, d_out_pointer, this->GetRows(), this->GetColumns(), matrix.GetRows(), matrix.GetColumns());

	hipDeviceSynchronize();

	h_outMatrix = d_outMatrix;
	outMatrix.ChangeAllValuesInMatrix(h_outMatrix);

	return outMatrix;
}
#pragma region RESZTA



Matrix Matrix::operator%(Matrix matrix) {
	Matrix outMatrix(this->GetRows(), this->GetColumns());

	host_vector<float> h_inMatrixA = this->GetAllMatrixValues();
	host_vector<float> h_inMatrixB = matrix.GetAllMatrixValues();
	host_vector<float> h_outMatrix = outMatrix.GetAllMatrixValues();
	device_vector<float> d_inMatrixA = h_inMatrixA;
	device_vector<float> d_inMatrixB = h_inMatrixB;
	device_vector<float> d_outMatrix = h_outMatrix;

	float *d_in_pointerA = raw_pointer_cast(d_inMatrixA.data());
	float *d_in_pointerB = raw_pointer_cast(d_inMatrixB.data());
	float *d_out_pointer = raw_pointer_cast(d_outMatrix.data());

	int blocks = this->GetRows() / NUMBER_OF_THREADS;
	blocks += 1;

	cudaMultiplication << <blocks, NUMBER_OF_THREADS >> > (d_in_pointerA, d_in_pointerB, d_out_pointer, this->GetRows(), this->GetColumns());

	hipDeviceSynchronize();

	h_outMatrix = d_outMatrix;
	outMatrix.ChangeAllValuesInMatrix(h_outMatrix);
	return outMatrix;
}
Matrix Matrix::operator+(Matrix matrix) {
	Matrix outMatrix(this->GetRows(), this->GetColumns());

	host_vector<float> h_inMatrixA = this->GetAllMatrixValues();
	host_vector<float> h_inMatrixB = matrix.GetAllMatrixValues();
	host_vector<float> h_outMatrix = outMatrix.GetAllMatrixValues();
	device_vector<float> d_inMatrixA = h_inMatrixA;
	device_vector<float> d_inMatrixB = h_inMatrixB;
	device_vector<float> d_outMatrix = h_outMatrix;

	float *d_in_pointerA = raw_pointer_cast(d_inMatrixA.data());
	float *d_in_pointerB = raw_pointer_cast(d_inMatrixB.data());
	float *d_out_pointer = raw_pointer_cast(d_outMatrix.data());

	int blocks = this->GetRows() / NUMBER_OF_THREADS;
	blocks += 1;

	cudaAddition << <blocks, NUMBER_OF_THREADS >> > (d_in_pointerA, d_in_pointerB, d_out_pointer, this->GetRows(), this->GetColumns());

	hipDeviceSynchronize();

	h_outMatrix = d_outMatrix;
	outMatrix.ChangeAllValuesInMatrix(h_outMatrix);
	return outMatrix;
}
Matrix Matrix::operator+(float value) {
	Matrix outMatrix(this->GetRows(), this->GetColumns());

	host_vector<float> h_inMatrixValues = this->GetAllMatrixValues();
	host_vector<float> h_outMatrixValues;
	device_vector<float> d_inMatrixValues = h_inMatrixValues;
	device_vector<float> d_outMatrixValues = h_inMatrixValues;

	float *d_in_pointer = raw_pointer_cast(d_inMatrixValues.data());
	float *d_out_pointer = raw_pointer_cast(d_outMatrixValues.data());

	int blocks = this->GetRows() / NUMBER_OF_THREADS;
	blocks += 1;

	cudaAdditionScalar << <blocks, NUMBER_OF_THREADS >> >(d_in_pointer, d_out_pointer, value, this->GetRows(), this->GetColumns());

	hipDeviceSynchronize();

	h_outMatrixValues = d_outMatrixValues;
	outMatrix.ChangeAllValuesInMatrix(h_outMatrixValues);

	return outMatrix;
}
Matrix Matrix::operator-(Matrix matrix) {
	Matrix outMatrix(this->GetRows(), this->GetColumns());

	host_vector<float> h_inMatrixA = this->GetAllMatrixValues();
	host_vector<float> h_inMatrixB = matrix.GetAllMatrixValues();
	host_vector<float> h_outMatrix = outMatrix.GetAllMatrixValues();
	device_vector<float> d_inMatrixA = h_inMatrixA;
	device_vector<float> d_inMatrixB = h_inMatrixB;
	device_vector<float> d_outMatrix = h_outMatrix;

	float *d_in_pointerA = raw_pointer_cast(d_inMatrixA.data());
	float *d_in_pointerB = raw_pointer_cast(d_inMatrixB.data());
	float *d_out_pointer = raw_pointer_cast(d_outMatrix.data());

	int blocks = this->GetRows() / NUMBER_OF_THREADS;
	blocks += 1;

	cudaSubtraction << <blocks, NUMBER_OF_THREADS >> > (d_in_pointerA, d_in_pointerB, d_out_pointer, this->GetRows(), this->GetColumns());

	hipDeviceSynchronize();

	h_outMatrix = d_outMatrix;
	outMatrix.ChangeAllValuesInMatrix(h_outMatrix);
	return outMatrix;
}
int Matrix::GetColumns() {
	return numberOfColumns;
}
int Matrix::GetRows() {
	return numberOfRows;
}
int Matrix::GetElementsCount() {
	return numberOfElements;
}
void Matrix::ChangeValueInMatrix(int row, int column, float value) {
	this->matrixValues[row*column + column] = value;
}
float Matrix::GetMatrixValue(int row, int column) {
	return this->matrixValues[row*this->GetColumns() + column];
}
host_vector<float> Matrix::GetAllMatrixValues() {
	return this->matrixValues;
}
void Matrix::SetColumns(int columns) {
	this->numberOfColumns = columns;
	this->SetElementsCount();
}
void Matrix::SetRows(int rows) {
	this->numberOfRows = rows;
	this->SetElementsCount();
}
void Matrix::ChangeAllValuesInMatrix(host_vector<float> &values) {
	this->matrixValues = values;
}
void Matrix::AttributeValuesToMatrix(host_vector<float> values) {
	if (this->handler.CheckIfThereAreTooManyValuesInMatrix(this, values) == this->handler.GetNoError())
		for (int i = 0; i < this->GetElementsCount(); i++)
		{
			this->matrixValues.push_back(values[i]);
		}
	else
	{
		AttributeZerosToMatrix();
	}
}
void Matrix::AttributeZerosToMatrix() {
	for (int i = 0; i < this->GetElementsCount(); i++)
	{
		this->matrixValues.push_back(0);
	}
}
void Matrix::AttributeOnesToMatrix() {
	for (int i = 0; i < this->GetElementsCount(); i++)
	{
		this->matrixValues.push_back(1);
	}
}
void Matrix::SetElementsCount() {
	this->numberOfElements = this->numberOfColumns * this->numberOfRows;
}
void Matrix::PrintMatrix() {
	for (int i = 0; i < this->GetRows(); i++)
	{
		for (int j = 0; j < this->GetColumns(); j++)
		{
			cout << this->GetMatrixValue(i, j) << "\t";
		}
		cout << endl;
	}
}

void Matrix::TransposeMatrixCPU(){
	host_vector<float> temp = this->GetAllMatrixValues();
	host_vector<float> outMatrix = temp;

	int rows = this->GetRows();
	int columns = this->GetColumns();

	for (int i = 0; i < rows; i++)
	{
		for (int k = 0; k < columns; k++)                                                                                                   
		{                                                        
			outMatrix[rows * k + i] = temp[i * columns + k];
		}
	}

	this->SetColumns(rows);
	this->SetRows(columns);
	this->ChangeAllValuesInMatrix(outMatrix);
}

Matrix Matrix::TransposeMatrixGPU(){
	device_vector<float> inMatrix = this->GetAllMatrixValues();
	device_vector<float> outMatrix = inMatrix;

	float* d_in_pointer = raw_pointer_cast(inMatrix.data());
	float* d_out_pointer = raw_pointer_cast(outMatrix.data());

	int numberOfElements = this->GetElementsCount();

	int rows = this->GetRows();
	int columns = this->GetColumns();

	int blocks = columns / NUMBER_OF_THREADS;
	blocks += 1;
	int threads = columns / blocks;
	threads += 1;

	cudaTranspose << < blocks, threads >> > (d_in_pointer, d_out_pointer, rows, columns);
	hipDeviceSynchronize();

	host_vector<float> transValues = outMatrix;

	Matrix newMatrix = Matrix(columns, rows, transValues);

	return newMatrix;
}

void Matrix::TransposeMatrixGPU_Shared() {
	device_vector<float> inMatrix = this->GetAllMatrixValues();
	device_vector<float> outMatrix = inMatrix;

	float* d_in_pointer = raw_pointer_cast(inMatrix.data());
	float* d_out_pointer = raw_pointer_cast(outMatrix.data());

	int rows = this->GetRows();
	int columns = this->GetColumns();

	int blocks = columns / NUMBER_OF_THREADS;
	blocks += 1;
	int threads = columns / blocks;
	threads += 1;

	cudaTransposeShared << < blocks, threads, (2 * columns * sizeof(float) + 4) >> >(d_in_pointer, d_out_pointer, rows, columns);
	hipDeviceSynchronize();

	host_vector<float> transValues = outMatrix;

	this->SetColumns(rows);
	this->SetRows(columns);
	this->ChangeAllValuesInMatrix(transValues);
}

float Matrix::SumAllElements()
{
	device_vector<float> matrix = this->GetAllMatrixValues();

	int rows = this->GetRows();
	int columns = this->GetColumns();

	vector<float> temp(rows, 0);
	host_vector<float> h_temp = temp;
	vector<float>().swap(temp);
	device_vector<float> sum = h_temp;

	//cout << h_temp[1];

	float* matrix_pointer = raw_pointer_cast(matrix.data());
	float* values_pointer = raw_pointer_cast(sum.data());

	int blocks = this->GetRows() / NUMBER_OF_THREADS;
	blocks += 1;

	cudaSumAll << < blocks, NUMBER_OF_THREADS >> > (matrix_pointer, rows, columns, values_pointer);
	hipDeviceSynchronize();

	h_temp = sum;
	float result = h_temp[0];

	return result;
}

void Matrix::PrintRow(int row, int start, int end) {
	if (end == -1)
	{
		end = this->GetRows()-1;
	}

	for (int i = start; i <= end; i++)
	{
		cout << this->GetMatrixValue(row, i) << "\t";
	}
	cout << endl;
}

void Matrix::PrintColumn(int column, int start, int end) {
	if (end == -1)
	{
		end = this->GetColumns() - 1;
	}

	for (int i = start; i <= end; i++)
	{
		cout << this->GetMatrixValue(i, column) << endl;
	}
}

#pragma endregion

__global__ void cudaMultiplicationScalar(float* inMatrixValues, float* outMatrixValues, float value, int matrixRows, int matrixColumns) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index < matrixRows)
	{
		for (int i = index * matrixColumns; i < (index + 1) * matrixColumns; i++)
		{
			outMatrixValues[i] = inMatrixValues[i] * value;
		}
	}

	return;
}

__global__ void cudaMultiplicationMatrix(float* inMatrixA, float* inMatrixB, float* outMatrix, int rowsA, int columnsA, int rowsB, int columnsB) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index < rowsA)
	{
		for (int i = index * columnsB; i < (index + 1) * columnsB; i++)
		{
			float value = 0;
			for (int k = index * columnsA, int m = i % columnsB; k < columnsA * (index + 1); k++, m += columnsB)
			{
				value = value + inMatrixA[k] * inMatrixB[m];
			}
			outMatrix[i] = value;
		}
	}
	return;
}
__global__ void cudaMultiplicationMatrix2(float* inMatrixA, float* inMatrixB, float* outMatrix, int rowsA, int columnsA, int rowsB, int columnsB) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if(rowsA*columnsB>index)
	{ 
	// rowsA X columnsB
	outMatrix[index] = 0;

	int row_index = index / columnsB;
	//rows_index = index - rows_index * columnsB;
	int column_index = index -(row_index*columnsB);
	//colums_index = index - colums_index * rowsA;
	int MatrixA_indeks = row_index * columnsA;
	int MatrixB_indeks =column_index;


	for (int i = 0; i < columnsA; i++)
		outMatrix[index]+=inMatrixA[MatrixA_indeks + i] * inMatrixB[MatrixB_indeks + columnsB * i];
	}
	return;
}

__global__ void cudaTranspose(float* inMatrixValues, float* outMatrixValues, int rows, int columns) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int x = (rows - 1)*columns;

	if (index < columns)
	{
		for (int i = index, int k = 0, int m = 0; i <= x + index; i += columns, m += 1)
		{
			outMatrixValues[m + index * rows] = inMatrixValues[i];

			if (k == 1) k = 0;
			else k++;
		}
	}
	return;
}

__global__ void cudaTransposeShared(float* inMatrixValues, float* outMatrixValues, int rows, int columns) {
	extern __shared__ short int temp[];

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int x = (rows - 1)*columns;

	if (index < columns)
	{
		for(int i = index, int k = 0, int m = 0; i <= x + index; i += columns, m+=1)
		{
			temp[k + 2*index] = inMatrixValues[i];
			outMatrixValues[m + index * rows] = temp[k + 2*index];
			
			if (k == 1) k = 0;
			else k++;
		}
	}
	return;
}

__global__ void cudaAddition(float* inMatrixA, float* inMatrixB, float* outMatrix, int rows, int columns) {

	int index = threadIdx.x + blockDim.x * blockIdx.x;

	if (index < rows)
	{
		for (int i = index * columns; i < (index + 1) * columns; i++)
		{
			outMatrix[i] = inMatrixA[i] + inMatrixB[i];
		}
	}
	return;
}

__global__ void cudaSubtraction(float* inMatrixA, float* inMatrixB, float* outMatrix, int rows, int columns) {

	int index = threadIdx.x + blockDim.x * blockIdx.x;

	if (index < rows)
	{
		for (int i = index * columns; i < (index + 1) * columns; i++)
		{
			outMatrix[i] = inMatrixA[i] - inMatrixB[i];
		}
	}
	return;
}

__global__ void cudaSumAll(float* matrix, int rows, int columns, float* outValue) {

	int index = threadIdx.x + blockDim.x * blockIdx.x;

	if (index < rows)
	{
		for (int i = index * columns; i < columns * (index + 1); i++)
		{
			outValue[index] = outValue[index] + matrix[i];
		}
	}

	__syncthreads();

	if (index == 0)
	{
		float buffer;

		for (int i = index; i < rows; i++)
		{
			buffer = buffer + outValue[i];
		}

		outValue[index] = buffer;
	}

	return;
}

__global__ void cudaAdditionScalar(float* inMatrixValues, float* outMatrixValues, float value, int matrixRows, int matrixColumns)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index < matrixRows)
	{
		for (int i = index * matrixColumns; i < (index + 1) * matrixColumns; i++)
		{
			outMatrixValues[i] = inMatrixValues[i] + value;
		}
	}

	return;
}
__global__ void cudaMultiplication(float* inMatrixA, float* inMatrixB, float* outMatrix, int rows, int columns)
{
	int index = threadIdx.x + blockDim.x * blockIdx.x;

	if (index < rows)
	{
		for (int i = index * columns; i < (index + 1) * columns; i++)
		{
			outMatrix[i] = inMatrixA[i] * inMatrixB[i];
		}
	}
	return;
}